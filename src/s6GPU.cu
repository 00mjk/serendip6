#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;
#include <stdexcept>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/gather.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>

#include "s6GPU.h"
#include "stopwatch.hpp"

//#define USE_TIMER
//#define USE_TOTAL_GPU_TIMER
#ifdef USE_TIMER
    bool use_timer=true;
#else
    bool use_timer=false;
#endif
#ifdef USE_TOTAL_GPU_TIMER
    bool use_total_gpu_timer=true;
#else
    bool use_total_gpu_timer=false;
#endif


//int init_device_vectors(int n_element, int n_input, device_vectors_t &dv) {
device_vectors_t * init_device_vectors(int n_element_max, int n_element_utilized, int n_input) {

    device_vectors_t * dv_p  = new device_vectors_t;

    dv_p->raw_timeseries_p   = new thrust::device_vector<char2>(n_element_max*n_input);
#ifdef TRANSPOSE
    dv_p->raw_timeseries_rowmaj_p   = new thrust::device_vector<char2>(n_element_max*n_input);
#endif
    dv_p->hit_indices_p      = new thrust::device_vector<int>();
    dv_p->hit_powers_p       = new thrust::device_vector<float>;
    dv_p->hit_baselines_p    = new thrust::device_vector<float>;

    return dv_p;
}

int init_device(int gpu_dev) {
    int rv = hipSetDevice(gpu_dev);
    // TODO error checking
    return rv;
}

void delete_device_vectors( device_vectors_t * dv_p) {
// TODO - is the right way to deallocate thrust vectors?
    delete(dv_p->raw_timeseries_p);
#ifdef TRANSPOSE
    delete(dv_p->raw_timeseries_rowmaj_p);
#endif
    delete(dv_p->hit_indices_p);      
    delete(dv_p->hit_powers_p);       
    delete(dv_p->hit_baselines_p);    

    delete(dv_p);
}

void create_fft_plan_1d_c2c(hipfftHandle* plan,
                            int          istride,
                            int          idist,
                            int          ostride,
                            int          odist,
                            size_t       nfft_,
                            size_t       nbatch) {
    int rank      = 1;
    int nfft[]    = {nfft_};
    int inembed[] = {nfft[0]};
    //int idist     = inembed[0];
    int onembed[] = {nfft[0]};
    //int odist     = onembed[0];
    hipfftResult fft_ret = hipfftPlanMany(plan,
                                        rank, nfft,
                                        inembed, istride, idist,
                                        onembed, ostride, odist,
                                        HIPFFT_C2C, nbatch);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftPlanMany failed");
    }
}

// Note: input == output is ok
void execute_fft_plan_c2c(hipfftHandle   *plan,
                          const float2* input,
                          float2*       output) {
    hipfftResult fft_ret = hipfftExecC2C(*plan,
                                       (hipfftComplex*)input,
                                       (hipfftComplex*)output,
                                       HIPFFT_FORWARD);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftExecC2C failed");
    }
}

// Functors
// --------
struct convert_complex_8b_to_float
    : public thrust::unary_function<char2,float2> {
    inline __host__ __device__
    float2 operator()(char2 a) const {
        return make_float2(a.x, a.y);
    }
};
struct compute_complex_power
    : public thrust::unary_function<float2,float> {
    inline __host__ __device__
    float operator()(float2 a) const {
        return a.x*a.x + a.y*a.y;
    }
};
struct advance_within_region
    : public thrust::unary_function<int,int> {
    int  delta;
    uint region_size;
    advance_within_region(int delta_, uint region_size_)
        : delta(delta_), region_size(region_size_) {}
    inline __host__ __device__
    int operator()(int i) const {
        int region = i / region_size;
        int idx    = i % region_size;
        idx += delta;
        idx = max(0, idx);
        idx = min(region_size-1, idx);
        return idx + region_size*region;
    }
};
struct running_mean_by_region
    : public thrust::unary_function<int, float> {
    uint         radius;
    uint         region_size;
    const float* d_scanned;
    running_mean_by_region(uint radius_,
                           uint region_size_,
                           const float* d_scanned_)
        : radius(radius_),
          region_size(region_size_),
          d_scanned(d_scanned_) {}
    inline __host__ __device__
    float operator()(uint i) const {
        uint region = i / region_size;
        uint offset = region * region_size;
        uint idx    = i % region_size;

        float sum;
        if( idx < radius ) {
            sum = (d_scanned[2*radius + offset] -
                   d_scanned[0 + offset]);
        }
        else if( idx > region_size-1-radius ) {
            sum = (d_scanned[region_size-1 + offset] -
                   d_scanned[region_size-1-2*radius + offset]);
        }
        else {
            sum = (d_scanned[idx + radius + offset] -
                   d_scanned[idx - radius + offset]);
        }
        return sum / (2*radius);
    }
};
struct transpose_index : public thrust::unary_function<size_t,size_t> {
// convert a linear index to a linear index in the transpose 
  size_t m, n;

  __host__ __device__
  transpose_index(size_t _m, size_t _n) : m(_m), n(_n) {}

  __host__ __device__
  size_t operator()(size_t linear_index)
  {
      size_t i = linear_index / n;
      size_t j = linear_index % n;

      return m * j + i;
  }
};
// --------
template<typename T>
struct divide_by : public thrust::unary_function<T,T> {
    T val;
    divide_by(T val_) : val(val_) {}
    inline __host__ __device__
    T operator()(T x) const {
        return x / val;
    }
};
template<typename T>
struct greater_than_val : public thrust::unary_function<T,bool> {
    T val;
    greater_than_val(T val_) : val(val_) {}
    inline __host__ __device__
    bool operator()(T x) const {
        return x > val;
    }
};
template <typename T>
void transpose(size_t m, size_t n, thrust::device_vector<T> *src, thrust::device_vector<T> *dst) {
// transpose an m-by-n array
  thrust::counting_iterator<size_t> indices(0);
  
  thrust::gather
    (thrust::make_transform_iterator(indices, transpose_index(n, m)),
     thrust::make_transform_iterator(indices, transpose_index(n, m)) + dst->size(),
     src->begin(),
     dst->begin());
}

void do_fft(hipfftHandle *fft_plan, float2* &fft_input_ptr, float2* &fft_output_ptr) {
    Stopwatch timer;
    if(use_timer) timer.start();
    execute_fft_plan_c2c(fft_plan, fft_input_ptr, fft_output_ptr);
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "FFT execution time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

void compute_power_spectrum(device_vectors_t *dv_p) {
    Stopwatch timer;
    if(use_timer) timer.start();
    thrust::transform(dv_p->fft_data_out_p->begin(), dv_p->fft_data_out_p->end(),
                      dv_p->powspec_p->begin(),
                      compute_complex_power());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Power spectrum time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

void compute_baseline(device_vectors_t *dv_p, int n_chan, int n_element, float smooth_scale) {
// Compute smoothed power spectrum baseline

    using thrust::make_transform_iterator;
    using thrust::make_counting_iterator;

    Stopwatch timer;
    if(use_timer) timer.start();
    thrust::exclusive_scan_by_key(make_transform_iterator(make_counting_iterator<int>(0),
                                                          //_1 / n_chan),
                                                          divide_by<int>(n_chan)),
                                  make_transform_iterator(make_counting_iterator<int>(n_element),
                                                          //_1 / n_chan),
                                                          divide_by<int>(n_chan)),
                                  dv_p->powspec_p->begin(),
                                  dv_p->scanned_p->begin());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Scan time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    if(use_timer) timer.start();
    const float* d_scanned_ptr = thrust::raw_pointer_cast(&(*dv_p->scanned_p)[0]);
  //const float* d_scanned_ptr = thrust::raw_pointer_cast(&(*dv.scanned_p   )[0]);
    thrust::transform(make_counting_iterator<uint>(0),
                      make_counting_iterator<uint>(n_element),
                      dv_p->baseline_p->begin(),
                      running_mean_by_region(smooth_scale,
                                             n_chan,
                                             d_scanned_ptr));
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Running mean time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

void normalize_power_spectrum(device_vectors_t *dv_p) {

    Stopwatch timer;
    if(use_timer) timer.start();
    thrust::transform(dv_p->powspec_p->begin(), dv_p->powspec_p->end(),
                      dv_p->baseline_p->begin(),
                      dv_p->normalised_p->begin(),
                      //_1 / _2);
                      thrust::divides<float>());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Normalisation time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

size_t find_hits(device_vectors_t *dv_p, int n_element, size_t maxhits, float power_thresh) {
// Extract and retrieve values exceeding the threshold

    using thrust::make_counting_iterator;

    size_t nhits;

    Stopwatch timer;
    if(use_timer) timer.start();
    dv_p->hit_indices_p->resize(n_element); // Note: Upper limit on required storage TODO - is n_element being set right?
    nhits = thrust::copy_if(make_counting_iterator<int>(0),
                                   make_counting_iterator<int>(n_element),
                                   dv_p->normalised_p->begin(),  // stencil
                                   dv_p->hit_indices_p->begin(), // result
                                   //_1 > power_thresh) - dv_p->hit_indices_p->begin();
                                   greater_than_val<float>(power_thresh))
                                                          - dv_p->hit_indices_p->begin();

    nhits = nhits > maxhits ? maxhits : nhits;       // overrun protection - hits beyond maxgpuhits are thrown away
    dv_p->hit_indices_p->resize(nhits);                 // this will only be resized downwards
                                            
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Hit extraction time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    if(use_timer) timer.start();
    // Retrieve hit info
    dv_p->hit_powers_p->resize(nhits);
    thrust::gather(dv_p->hit_indices_p->begin(), dv_p->hit_indices_p->end(),
                   dv_p->powspec_p->begin(),
                   dv_p->hit_powers_p->begin());
    dv_p->hit_baselines_p->resize(nhits);
    thrust::gather(dv_p->hit_indices_p->begin(), dv_p->hit_indices_p->end(),
                   dv_p->baseline_p->begin(),
                   dv_p->hit_baselines_p->begin());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Hit info gather time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

    return nhits;
}    

// AO spectra order goes as pol0chan0 pol0chan1    pol1chan0 pol1chan1    pol0chan2 pol0chan3    pol1chan2 pol1chan3... 
inline int ao_pol(long spectrum_index) {
    return((long)floor((double)spectrum_index/2) % 2);
}
inline int ao_coarse_chan(long spectrum_index) {
    return((long)floor((double)spectrum_index/4) * 2 + spectrum_index % 2);
}
    
int spectroscopy(int n_subband,
                 int n_chan,
                 int n_input,
                 int beam,
                 size_t maxhits,
                 size_t maxgpuhits,
                 float power_thresh,
                 float smooth_scale,
                 uint64_t * input_data,
                 size_t n_input_data_bytes,
                 s6_output_block_t *s6_output_block,
                 device_vectors_t    *dv_p,
                 hipfftHandle *fft_plan) {

// GPU memory allocation note.  Our total memory needs are larger than the
// capcity of our current GPU (GeForce GTX 780 Ti with 3071MB). So we allocate 
// as needed and delete memory as soon as it is no longer needed.

    Stopwatch timer; 
    Stopwatch total_gpu_timer;
    int n_element = n_subband*n_chan*n_input;
    size_t nhits;
    //size_t prior_nhits=0;
    size_t total_nhits=0;

    char2 * h_raw_timeseries = (char2 *)input_data;

    if(use_total_gpu_timer) total_gpu_timer.start();

    // Copy to the device
    if(use_timer) timer.start();
    thrust::copy(h_raw_timeseries, h_raw_timeseries + n_input_data_bytes / sizeof(char2),
                 //d_raw_timeseries.begin());
                 dv_p->raw_timeseries_p->begin());
    if(use_timer) timer.stop();
    if(use_timer) cout << "H2D time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

    // allocate GPU memory for the FFTs and power spectra
    dv_p->fft_data_p         = new thrust::device_vector<float2>(n_element);
    dv_p->fft_data_out_p     = new thrust::device_vector<float2>(n_element);
    dv_p->powspec_p          = new thrust::device_vector<float>(n_element);

    if(use_timer) timer.start();
    // Unpack from 8-bit to floats
    thrust::transform(dv_p->raw_timeseries_p->begin(), 
                      dv_p->raw_timeseries_p->end(),
                      dv_p->fft_data_p->begin(),
                      convert_complex_8b_to_float());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Unpack time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    // input pointer varies with input
    float2* fft_input_ptr  = thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[0]));
    // output pointer is constant - we reuse the output area for each input
    float2* fft_output_ptr = thrust::raw_pointer_cast(&((*dv_p->fft_data_out_p)[0]));
    //fprintf(stderr, "fft_input_ptr = %p  fft_output_ptr = %p\n", fft_input_ptr, fft_output_ptr);

    //fprintf(stderr, "n_chan %d n_element %d maxhits %d smooth_scale %f power_thresh %f\n", 
    //        n_chan, n_element, maxhits, smooth_scale, power_thresh);

    do_fft                      (fft_plan, fft_input_ptr, fft_output_ptr);
    compute_power_spectrum      (dv_p);

    // done with the FFTs - delete the associated GPU memory
    delete(dv_p->fft_data_p);         
    delete(dv_p->fft_data_out_p);     
    // and allocate GPU memory for power normalization
    dv_p->scanned_p          = new thrust::device_vector<float>(n_element);
    dv_p->baseline_p         = new thrust::device_vector<float>(n_element);
    dv_p->normalised_p       = new thrust::device_vector<float>(n_element);

    compute_baseline            (dv_p, n_chan, n_element, smooth_scale);
    normalize_power_spectrum    (dv_p);
    nhits = find_hits           (dv_p, n_element, maxhits, power_thresh);
    // TODO should probably report if nhits == maxgpuhits, ie overflow
    
    // copy to return vector
    nhits = nhits > maxhits ? maxhits : nhits;
    if(use_timer) timer.start();

    total_nhits += nhits;
    s6_output_block->header.nhits[beam] = nhits;
    thrust::copy(dv_p->hit_powers_p->begin(),    dv_p->hit_powers_p->end(),    &s6_output_block->power[beam][0]);
    thrust::copy(dv_p->hit_baselines_p->begin(), dv_p->hit_baselines_p->end(), &s6_output_block->baseline[beam][0]);
    thrust::copy(dv_p->hit_indices_p->begin(),   dv_p->hit_indices_p->end(),   &s6_output_block->hit_indices[beam][0]);
    for(size_t i=0; i<nhits; ++i) {
        long hit_index                        = s6_output_block->hit_indices[beam][i]; 
        long spectrum_index                   = (long)floor((double)hit_index/n_chan);
        s6_output_block->pol[beam][i]         = ao_pol(spectrum_index);
        s6_output_block->coarse_chan[beam][i] = ao_coarse_chan(spectrum_index);
        s6_output_block->fine_chan[beam][i]   = hit_index % n_chan;
        //fprintf(stderr, "hit_index %ld spectrum_index %ld pol %d cchan %d fchan %d power %f\n", 
        //        hit_index, spectrum_index, s6_output_block->pol[beam][i], s6_output_block->coarse_chan[beam][i], 
        //        s6_output_block->fine_chan[beam][i], s6_output_block->power[beam][i]);
    }
        
    // delete remaining GPU memory
    delete(dv_p->powspec_p);          
    delete(dv_p->scanned_p);          
    delete(dv_p->baseline_p);         
    delete(dv_p->normalised_p);       
       
    if(use_timer) timer.stop();
    if(use_timer) cout << "Copy to return vector time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

    if(use_total_gpu_timer) total_gpu_timer.stop();
    if(use_total_gpu_timer) cout << "Total GPU time:\t" << total_gpu_timer.getTime() << endl;
    if(use_total_gpu_timer) total_gpu_timer.reset();
    
    return total_nhits;
}
