#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;
#include <stdexcept>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/gather.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>

#include "s6GPU.h"
#include "stopwatch.hpp"

//#define USE_TIMER
#define USE_TOTAL_GPU_TIMER
#ifdef USE_TIMER
    bool use_timer=true;
#else
    bool use_timer=false;
#endif
#ifdef USE_TOTAL_GPU_TIMER
    bool use_total_gpu_timer=true;
#else
    bool use_total_gpu_timer=false;
#endif


//int init_device_vectors(int n_element, int n_input, device_vectors_t &dv) {
device_vectors_t * init_device_vectors(int n_element, int n_input) {

    device_vectors_t * dv_p  = new device_vectors_t;

    dv_p->raw_timeseries_p   = new thrust::device_vector<char2>(n_element*n_input);
#ifdef TRANSPOSE
    dv_p->raw_timeseries_rowmaj_p   = new thrust::device_vector<char2>(n_element*n_input);
#endif
    dv_p->fft_data_p         = new thrust::device_vector<float2>(n_element*n_input);
    dv_p->fft_data_out_p     = new thrust::device_vector<float2>(n_element);
    dv_p->powspec_p          = new thrust::device_vector<float>(n_element);
    dv_p->scanned_p          = new thrust::device_vector<float>(n_element);
    dv_p->baseline_p         = new thrust::device_vector<float>(n_element);
    dv_p->normalised_p       = new thrust::device_vector<float>(n_element);
    dv_p->hit_indices_p      = new thrust::device_vector<int>();
    dv_p->hit_powers_p       = new thrust::device_vector<float>;
    dv_p->hit_baselines_p    = new thrust::device_vector<float>;
#ifdef COMPUTE_HIT_DENSITY
    dv_p->hit_indices_high_p = new thrust::device_vector<int>;
    dv_p->hit_indices_low_p  = new thrust::device_vector<int>;
    dv_p->hit_densities_p    = new thrust::device_vector<int>;
#endif

    return dv_p;
}

int init_device(int gpu_dev) {
    int rv = hipSetDevice(gpu_dev);
    // TODO error checking
    return rv;
}

void delete_device_vectors( device_vectors_t * dv_p) {
// TODO - is the right way to deallocate thrust vectors?
    delete(dv_p->raw_timeseries_p);
#ifdef TRANSPOSE
    delete(dv_p->raw_timeseries_rowmaj_p);
#endif
    delete(dv_p->fft_data_p);         
    delete(dv_p->fft_data_out_p);     
    delete(dv_p->powspec_p);          
    delete(dv_p->scanned_p);          
    delete(dv_p->baseline_p);         
    delete(dv_p->normalised_p);       
    delete(dv_p->hit_indices_p);      
    delete(dv_p->hit_powers_p);       
    delete(dv_p->hit_baselines_p);    
#ifdef COMPUTE_HIT_DENSITY
    delete(dv_p->hit_indices_high_p); 
    delete(dv_p->hit_indices_low_p);  
    delete(dv_p->hit_densities_p);    
#endif

    delete(dv_p);
}

void create_fft_plan_1d_c2c(hipfftHandle* plan,
                            int          istride,
                            int          idist,
                            int          ostride,
                            int          odist,
                            size_t       nfft_,
                            size_t       nbatch) {
    int rank      = 1;
    int nfft[]    = {nfft_};
    int inembed[] = {nfft[0]};
    //int idist     = inembed[0];
    int onembed[] = {nfft[0]};
    //int odist     = onembed[0];
    hipfftResult fft_ret = hipfftPlanMany(plan,
                                        rank, nfft,
                                        inembed, istride, idist,
                                        onembed, ostride, odist,
                                        HIPFFT_C2C, nbatch);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftPlanMany failed");
    }
}

// Note: input == output is ok
void execute_fft_plan_c2c(hipfftHandle   *plan,
                          const float2* input,
                          float2*       output) {
    hipfftResult fft_ret = hipfftExecC2C(*plan,
                                       (hipfftComplex*)input,
                                       (hipfftComplex*)output,
                                       HIPFFT_FORWARD);
    if( fft_ret != HIPFFT_SUCCESS ) {
        throw std::runtime_error("hipfftExecC2C failed");
    }
}

// Functors
// --------
struct convert_complex_8b_to_float
    : public thrust::unary_function<char2,float2> {
    inline __host__ __device__
    float2 operator()(char2 a) const {
        return make_float2(a.x, a.y);
    }
};
struct compute_complex_power
    : public thrust::unary_function<float2,float> {
    inline __host__ __device__
    float operator()(float2 a) const {
        return a.x*a.x + a.y*a.y;
    }
};
struct advance_within_region
    : public thrust::unary_function<int,int> {
    int  delta;
    uint region_size;
    advance_within_region(int delta_, uint region_size_)
        : delta(delta_), region_size(region_size_) {}
    inline __host__ __device__
    int operator()(int i) const {
        int region = i / region_size;
        int idx    = i % region_size;
        idx += delta;
        idx = max(0, idx);
        idx = min(region_size-1, idx);
        return idx + region_size*region;
    }
};
struct running_mean_by_region
    : public thrust::unary_function<int, float> {
    uint         radius;
    uint         region_size;
    const float* d_scanned;
    running_mean_by_region(uint radius_,
                           uint region_size_,
                           const float* d_scanned_)
        : radius(radius_),
          region_size(region_size_),
          d_scanned(d_scanned_) {}
    inline __host__ __device__
    float operator()(uint i) const {
        uint region = i / region_size;
        uint offset = region * region_size;
        uint idx    = i % region_size;

        float sum;
        if( idx < radius ) {
            sum = (d_scanned[2*radius + offset] -
                   d_scanned[0 + offset]);
        }
        else if( idx > region_size-1-radius ) {
            sum = (d_scanned[region_size-1 + offset] -
                   d_scanned[region_size-1-2*radius + offset]);
        }
        else {
            sum = (d_scanned[idx + radius + offset] -
                   d_scanned[idx - radius + offset]);
        }
        return sum / (2*radius);
    }
};
struct transpose_index : public thrust::unary_function<size_t,size_t> {
// convert a linear index to a linear index in the transpose 
  size_t m, n;

  __host__ __device__
  transpose_index(size_t _m, size_t _n) : m(_m), n(_n) {}

  __host__ __device__
  size_t operator()(size_t linear_index)
  {
      size_t i = linear_index / n;
      size_t j = linear_index % n;

      return m * j + i;
  }
};
// --------
template<typename T>
struct divide_by : public thrust::unary_function<T,T> {
    T val;
    divide_by(T val_) : val(val_) {}
    inline __host__ __device__
    T operator()(T x) const {
        return x / val;
    }
};
template<typename T>
struct greater_than_val : public thrust::unary_function<T,bool> {
    T val;
    greater_than_val(T val_) : val(val_) {}
    inline __host__ __device__
    bool operator()(T x) const {
        return x > val;
    }
};
template <typename T>
void transpose(size_t m, size_t n, thrust::device_vector<T> *src, thrust::device_vector<T> *dst) {
// transpose an m-by-n array
  thrust::counting_iterator<size_t> indices(0);
  
  thrust::gather
    (thrust::make_transform_iterator(indices, transpose_index(n, m)),
     thrust::make_transform_iterator(indices, transpose_index(n, m)) + dst->size(),
     src->begin(),
     dst->begin());
}

void do_fft(hipfftHandle *fft_plan, float2* &fft_input_ptr, float2* &fft_output_ptr) {
    Stopwatch timer;
    if(use_timer) timer.start();
    execute_fft_plan_c2c(fft_plan, fft_input_ptr, fft_output_ptr);
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "FFT execution time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

void compute_power_spectrum(device_vectors_t *dv_p) {
    Stopwatch timer;
    if(use_timer) timer.start();
    thrust::transform(dv_p->fft_data_out_p->begin(), dv_p->fft_data_out_p->end(),
                      dv_p->powspec_p->begin(),
                      compute_complex_power());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Power spectrum time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

void compute_baseline(device_vectors_t *dv_p, int n_chan, int n_element, float smooth_scale) {
// Compute smoothed power spectrum baseline

    using thrust::make_transform_iterator;
    using thrust::make_counting_iterator;

    Stopwatch timer;
    if(use_timer) timer.start();
    thrust::exclusive_scan_by_key(make_transform_iterator(make_counting_iterator<int>(0),
                                                          //_1 / n_chan),
                                                          divide_by<int>(n_chan)),
                                  make_transform_iterator(make_counting_iterator<int>(n_element),
                                                          //_1 / n_chan),
                                                          divide_by<int>(n_chan)),
                                  dv_p->powspec_p->begin(),
                                  dv_p->scanned_p->begin());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Scan time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    if(use_timer) timer.start();
    const float* d_scanned_ptr = thrust::raw_pointer_cast(&(*dv_p->scanned_p)[0]);
  //const float* d_scanned_ptr = thrust::raw_pointer_cast(&(*dv.scanned_p   )[0]);
    thrust::transform(make_counting_iterator<uint>(0),
                      make_counting_iterator<uint>(n_element),
                      dv_p->baseline_p->begin(),
                      running_mean_by_region(smooth_scale,
                                             n_chan,
                                             d_scanned_ptr));
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Running mean time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

void normalize_power_spectrum(device_vectors_t *dv_p) {

    Stopwatch timer;
    if(use_timer) timer.start();
    thrust::transform(dv_p->powspec_p->begin(), dv_p->powspec_p->end(),
                      dv_p->baseline_p->begin(),
                      dv_p->normalised_p->begin(),
                      //_1 / _2);
                      thrust::divides<float>());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Normalisation time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}

size_t find_hits(device_vectors_t *dv_p, int n_element, size_t maxgpuhits, float power_thresh) {
// Extract and retrieve values exceeding the threshold

    using thrust::make_counting_iterator;

    size_t nhits;

    Stopwatch timer;
    if(use_timer) timer.start();
    dv_p->hit_indices_p->resize(n_element); // Note: Upper limit on required storage TODO - is n_element being set right?
    nhits = thrust::copy_if(make_counting_iterator<int>(0),
                                   make_counting_iterator<int>(n_element),
                                   dv_p->normalised_p->begin(),  // stencil
                                   dv_p->hit_indices_p->begin(), // result
                                   //_1 > power_thresh) - dv_p->hit_indices_p->begin();
                                   greater_than_val<float>(power_thresh))
                                                          - dv_p->hit_indices_p->begin();

    nhits = nhits > maxgpuhits ? maxgpuhits : nhits;    // overrun protection - hits beyond maxgpuhits are thrown away
    dv_p->hit_indices_p->resize(nhits);                 // this will only be resized downwards
                                            
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Hit extraction time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    if(use_timer) timer.start();
    // Retrieve hit info
    dv_p->hit_powers_p->resize(nhits);
    thrust::gather(dv_p->hit_indices_p->begin(), dv_p->hit_indices_p->end(),
                   dv_p->powspec_p->begin(),
                   dv_p->hit_powers_p->begin());
    dv_p->hit_baselines_p->resize(nhits);
    thrust::gather(dv_p->hit_indices_p->begin(), dv_p->hit_indices_p->end(),
                   dv_p->baseline_p->begin(),
                   dv_p->hit_baselines_p->begin());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Hit info gather time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

    return nhits;
}    

#ifdef COMPUTE_HIT_DENSITY
// This is not needed for production but is kept here for reference
void compute_hit_density(device_vectors_t &dv, size_t nhits, int n_chan, float smooth_scale) {
    using thrust::make_transform_iterator;

    Stopwatch timer;
    if(use_timer) timer.start();
    // Compute hit density
    // Note: This searches forward and backward smooth_scale/2 indices
    //         from each hit and counts how many hits lie in that range.
    dv.hit_indices_high_p->resize(nhits);
    int fwd = smooth_scale/2+1;
    int bck = smooth_scale/2;
    thrust::lower_bound(dv.hit_indices_p->begin(), dv.hit_indices_p->end(),
                        make_transform_iterator(dv.hit_indices_p->begin(),
                                                advance_within_region(fwd,
                                                                      n_chan)),
                        make_transform_iterator(dv.hit_indices_p->end(),
                                                advance_within_region(fwd,
                                                                      n_chan)),
                        dv.hit_indices_high_p->begin());
    dv.hit_indices_low_p->resize(nhits);
    thrust::upper_bound(dv.hit_indices_p->begin(), dv.hit_indices_p->end(),
                        make_transform_iterator(dv.hit_indices_p->begin(),
                                                advance_within_region(-bck,
                                                                      n_chan)),
                        make_transform_iterator(dv.hit_indices_p->end(),
                                                advance_within_region(-bck,
                                                                      n_chan)),
                        dv.hit_indices_low_p->begin());
    dv.hit_densities_p->resize(nhits);
    thrust::transform(dv.hit_indices_high_p->begin(), dv.hit_indices_high_p->end(),
                      dv.hit_indices_low_p->begin(),
                      dv.hit_densities_p->begin(),
                      //_1 - _2);
                      //thrust::minus<float>());
                      thrust::minus<int>());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Hit density time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
}    
#endif  //COMPUTE_HIT_DENSITY

int spectroscopy(int n_subband,
                 int n_chan,
                 int n_input,
                 int beam,
                 size_t maxhits,
                 size_t maxgpuhits,
                 float power_thresh,
                 float smooth_scale,
                 uint64_t * input_data,
                 size_t n_input_data_bytes,
                 hits_t *hits_p, 
                 device_vectors_t    *dv_p,
                 hipfftHandle *fft_plan) {

    Stopwatch timer;
    Stopwatch total_gpu_timer;
    int n_element = n_subband*n_chan;
    size_t nhits;
    size_t prior_nhits=0;
    size_t total_nhits=0;

    char2 * h_raw_timeseries = (char2 *)input_data;

    if(use_total_gpu_timer) total_gpu_timer.start();

    // Copy to the device
    if(use_timer) timer.start();
    thrust::copy(h_raw_timeseries, h_raw_timeseries + n_input_data_bytes / sizeof(char2),
                 //d_raw_timeseries.begin());
                 dv_p->raw_timeseries_p->begin());
    if(use_timer) timer.stop();
    if(use_timer) cout << "H2D time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();

    if(use_timer) timer.start();
    // Unpack from 8-bit to floats
    thrust::transform(dv_p->raw_timeseries_p->begin(), 
                      dv_p->raw_timeseries_p->end(),
                      dv_p->fft_data_p->begin(),
                      convert_complex_8b_to_float());
    hipDeviceSynchronize();
    if(use_timer) timer.stop();
    if(use_timer) cout << "Unpack time:\t" << timer.getTime() << endl;
    if(use_timer) timer.reset();
    
    for(int input=0; input<n_input; input++) {

        // input pointer varies with input
        float2* fft_input_ptr  = thrust::raw_pointer_cast(&((*dv_p->fft_data_p)[input]));
        // output pointer is constant - we reuse the output area for each input
        float2* fft_output_ptr = thrust::raw_pointer_cast(&((*dv_p->fft_data_out_p)[0]));
        //fprintf(stderr, "fft_input_ptr = %p  fft_output_ptr = %p\n", fft_input_ptr, fft_output_ptr);

        do_fft                      (fft_plan, fft_input_ptr, fft_output_ptr);
        compute_power_spectrum      (dv_p);
        compute_baseline            (dv_p, n_chan, n_element, smooth_scale);
        normalize_power_spectrum    (dv_p);
        nhits = find_hits           (dv_p, n_element, maxgpuhits, power_thresh);
        // TODO should probably report if nhits == maxgpuhits, ie overflow
#ifdef COMPUTE_HIT_DENSITY
// This is not needed for production but is kept here for reference
        //compute_hit_density         (dv, nhits, n_chan, smooth_scale);
#endif
    
        // copy to return vector
        nhits = nhits > maxhits ? maxhits : nhits;
        if(use_timer) timer.start();
        for(size_t i=prior_nhits, j=0; j<nhits; ++i, j++ ) {
            int idx               = (*(dv_p->hit_indices_p))[j];
            hits_p[i].power       = (*(dv_p->hit_powers_p))[j];
            hits_p[i].baseline    = (*(dv_p->hit_baselines_p))[j];
            hits_p[i].strength    = hits_p[j].power / hits_p[j].baseline;
#ifdef COMPUTE_HIT_DENSITY
// This is not needed for production but is kept here for reference
            //hits_p[i].density   = (*dv.hit_densities_p)[j]; 
#endif
            hits_p[i].coarse_chan = idx / n_chan;
            hits_p[i].fine_chan   = idx % n_chan;
            hits_p[i].input       = input;
            hits_p[i].beam        = beam;
        }
        prior_nhits = nhits;
        total_nhits += nhits;
        if(use_timer) timer.stop();
        if(use_timer) cout << "Copy to return vector time:\t" << timer.getTime() << endl;
        if(use_timer) timer.reset();
    }  // for each input

    if(use_total_gpu_timer) total_gpu_timer.stop();
    if(use_total_gpu_timer) cout << "Total GPU time:\t" << total_gpu_timer.getTime() << endl;
    if(use_total_gpu_timer) total_gpu_timer.reset();
    
    return total_nhits;
}
